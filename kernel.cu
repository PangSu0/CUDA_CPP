#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <cstdlib>
#include <ctime>
#include <iostream>
#include "run.h"
#include <conio.h>
#include <list>

using namespace std;
#define BLOCK_SIZE 10

typedef struct
{
	int height;
	int width;
	double* elements;
}Matrix;

__global__ void MatMulKernel(const Matrix, const Matrix, Matrix);

void MatMul(const Matrix A, const Matrix B, Matrix C)
{
	// Load A and B to device memory
	Matrix d_A;
	d_A.width = A.width; d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(double);
	hipMalloc(&d_A.elements, size);
	hipMemcpy(d_A.elements, A.elements, size,
		hipMemcpyHostToDevice);
	Matrix d_B;
	d_B.width = B.width; d_B.height = B.height;
	size = B.width * B.height * sizeof(double);
	hipMalloc(&d_B.elements, size);
	hipMemcpy(d_B.elements, B.elements, size,
		hipMemcpyHostToDevice);

	// Allocate C in device memory
	Matrix d_C;
	d_C.width = C.width; d_C.height = C.height;
	size = C.width * C.height * sizeof(double);
	hipMalloc(&d_C.elements, size);

	// Invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
	MatMulKernel << <dimGrid, dimBlock >> > (d_A, d_B, d_C);

	// Read C from device memory
	hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);

	// Free device memory
	hipFree(d_A.elements);
	hipFree(d_B.elements);
	hipFree(d_C.elements);
}

__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{
	// Each thread computes one element of C
	// by accumulating results into Cvalue
	float Cvalue = 0;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	for (int e = 0; e < A.width; ++e)
		Cvalue += A.elements[row * A.width + e]
		* B.elements[e * B.width + col];
	C.elements[row * C.width + col] = Cvalue;
}

double ReturnFromZeroToOneExclusives() // 0~1 ������ ��
{
	return (double)(rand()+1) / (RAND_MAX + 2);
}

int main()
{
	std::list<rotation> cubeFormula;
	CUBE cube(cubeFormula);
	cube.Setup();
	
	//srand((unsigned)time(NULL));
	/*srand(1);
	const int arraySizeY = 10;
	const int arraySizeM = 15;
	const int arraySizeX = 20;
	Matrix A;
	Matrix B;
	Matrix C;
	A.height = arraySizeY;
	A.width = arraySizeM;
	B.height = arraySizeM;
	B.width = arraySizeX;
	C.height = arraySizeY;
	C.width = arraySizeX;

	double a[arraySizeY * arraySizeM] = { 0 };
	double b[arraySizeM * arraySizeX] = { 0 };
	double c[arraySizeY * arraySizeX] = { 0 };

	for (int i = 0; i < arraySizeY * arraySizeM; i++)
		a[i] = ReturnFromZeroToOneExclusives();
	for (int i = 0; i < arraySizeM * arraySizeX; i++)
		b[i] = ReturnFromZeroToOneExclusives();

	A.elements = a;
	B.elements = b;
	C.elements = c;

	MatMul(A, B ,C);

	for (int i = 0; i < arraySizeY * arraySizeX; i++)
	{
		if (i % arraySizeY == 0)
			putchar('\n');
		printf("%6.10f\t", C.elements[i]);
	}
	putchar('\n');
	hipDeviceReset();*/

	return 0;
}
