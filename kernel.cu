#include "header.h"

int main()
{
	const int arraySizeY = 10;
	const int arraySizeM = 15;
	const int arraySizeX = 20;
	Matrix<int> A(10, 15);
	Matrix<int> B(15, 20);
	Matrix<int> C(10, 20);
	 //��� a,b,c �� �����.
	int a[arraySizeY * arraySizeM] = { 0 };
	int b[arraySizeM * arraySizeX] = { 0 };
	int c[arraySizeY * arraySizeX] = { 0 };

	// �˸��� ������ �ʱ�ȭ �Ѵ�.
	for (int i = 0; i < arraySizeY * arraySizeM; i++)
		a[i] = i;
	for (int i = 0; i < arraySizeM * arraySizeX; i++)
		b[i] = i;

	A.SetElements(a);
	B.SetElements(b);
	C.SetElements(c);

	// �۾��� �Լ��� ���Ѵ�.
	MatMul<int>(A, B ,C);

	// ����� ����Ѵ�.
	for (int i = 0; i < arraySizeY * arraySizeX; i++)
	{
		if (i % arraySizeY == 0)
			putchar('\n');
		printf("%d ", c[i]);
	}
	putchar('\n');
	hipDeviceReset();

	return 0;
}
